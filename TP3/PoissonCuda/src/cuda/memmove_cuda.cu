#include "memmove.hxx"
#include "cuda_check.cuh"

double * allocate(int n) {
  double *d;
  CUDA_CHECK_OP(hipMalloc(&d, n*sizeof(double)));
  return d;
}
  
void deallocate(double *&d) {
  CUDA_CHECK_OP(hipFree(d));
  d = NULL;
}
  
void copyDeviceToHost(double *h, double *d, int n)
{
  hipMemcpy(h, d, n * sizeof(double), hipMemcpyDeviceToHost);
}
  
void copyHostToDevice(double *h, double *d, int n)
{
  hipMemcpy(h, d, n * sizeof(double), hipMemcpyHostToDevice);
}
  
void copyDeviceToDevice(double *d_out, double *d_in, int n)
{
  hipMemcpy(d_out, d_in, n * sizeof(double), hipMemcpyDeviceToDevice);
}
  
  
