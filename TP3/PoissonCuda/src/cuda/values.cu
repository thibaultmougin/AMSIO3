#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>
#include <iomanip>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>

#include "cuda_check.cuh"
#include "dim.cuh"
#include "user.cuh"

#include "values.hxx"
#include "os.hxx"

__global__ void dprint(double *u)
{
  int i, j, k;
  int p;

  for (k=0; k<d_n[2]; k++) {
    for (j=0; j<d_n[1]; j++) {
      for (i=0; i<d_n[0]; i++) {
        p = i + d_n[0] * (j + d_n[1]*k);
        printf(" %f", u[p]);
      }
      printf("\n");
    }
    printf("\n");
  }
}

__global__
void zeroValue(double *u, int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i<n) {
    u[i] = 0.0;
  }
}

void zero(double *d, int n)
{
  int dimBlock = 256;
  int dimGrid = (n + dimBlock - 1)/dimBlock;

  zeroValue<<<dimGrid, dimBlock>>>(d, n);
  CUDA_CHECK_KERNEL();
}


__global__
void initValue(double *u)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  int p;

  if (i>0 && i<d_n[0]-1 && j>0 && j<d_n[1]-1 && k>0 && k<d_n[2]-1)
  {
    p = i + d_n[0] * (j + k*d_n[1]);
    u[p] = cond_ini(d_xmin[0] + i*d_dx[0],
                    d_xmin[1] + j*d_dx[1], 
                    d_xmin[2] + k*d_dx[2]);
  }
}

void init(double *d, int n[3])
{
  dim3 dimBlock(8,8,8);
  dim3 dimGrid(int(ceil(n[0]/double(dimBlock.x))),
               int(ceil(n[1]/double(dimBlock.y))),
               int(ceil(n[2]/double(dimBlock.z))));

  initValue<<<dimGrid, dimBlock>>>(d);
  hipDeviceSynchronize();
  CUDA_CHECK_KERNEL();
}

__global__
void boundZValue(int k, double *u)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int p;

  if (i<d_n[0] && j<d_n[1]) {
    p = i + j*d_n[0] + k*d_n[0]*d_n[1];
    u[p] = cond_ini(d_xmin[0] + i*d_dx[0], 
                    d_xmin[1] + j*d_dx[1], 
                    d_xmin[2] + k*d_dx[2]);    
  }
}


__global__
void boundYValue(int j, double *u)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  int p;

  if (i<d_n[0] && k<d_n[2]) {
    p = i + j*d_n[0] + k*d_n[0]*d_n[1];
    u[p] = cond_ini(d_xmin[0] + i*d_dx[0], 
                    d_xmin[1] + j*d_dx[1], 
                    d_xmin[2] + k*d_dx[2]);
  }
}

__global__
void boundXValue(int i, double *u)
{
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int k = threadIdx.z + blockIdx.z * blockDim.z;
  int p;

  if (j<d_n[1] && k<d_n[2]) {
    p = i + j*d_n[0] + k*d_n[0]*d_n[1];
    u[p] = cond_ini(d_xmin[0] + i*d_dx[0], 
                    d_xmin[1] + j*d_dx[1], 
                    d_xmin[2] + k*d_dx[2]);
  }
}

void boundaries(double *d, int n[3], int imin[3], int imax[3])
{
  dim3 dimBlock2(16,16,1);
  dim3 dimGrid2(int(ceil(n[0]/double(dimBlock2.x))),
                int(ceil(n[1]/double(dimBlock2.y))), 
                1);
  boundZValue<<<dimGrid2, dimBlock2>>>(imin[2]-1, d);

  boundZValue<<<dimGrid2, dimBlock2>>>(imax[2]+1, d);

  dim3 dimBlock1(16,1,16);
  dim3 dimGrid1(int(ceil(n[0]/double(dimBlock1.x))), 
                1,
                int(ceil(n[2]/double(dimBlock1.z))));

  boundYValue<<<dimGrid1, dimBlock1>>>(imin[1]-1, d);
  boundYValue<<<dimGrid1, dimBlock1>>>(imax[1]+1, d);

  dim3 dimBlock0(1,16,16);
  dim3 dimGrid0(1, 
                int(ceil(n[1]/double(dimBlock0.y))),
                int(ceil(n[2]/double(dimBlock0.z))));

  boundXValue<<<dimGrid0, dimBlock0>>>(imin[0]-1, d);
  boundXValue<<<dimGrid0, dimBlock0>>>(imax[0]+1, d);

  hipDeviceSynchronize();
  CUDA_CHECK_KERNEL();
}
