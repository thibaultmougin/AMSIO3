#include "hip/hip_runtime.h"
#include "dim.cuh"
#include "cuda_check.cuh"
#include "user.cuh"

#include "timer.hxx"
#include "iteration.hxx"

__device__
double cond_ini(double x, double y, double z){
  x -= 0.5;
  y -= 0.5;
  z -= 0.5;
  if (x * x + y * y + z * z < 0.1)
    return 1.0;
  else
    return 0.0;
}

__device__
double cond_lim(double x, double y, double z)
{
  return 0.0;
}

__device__
double force(double x, double y, double z)
{
  return sin(x - 0.5) * cos(y - 0.5) * exp(-z * z);
}


