#include "hip/hip_runtime.h"
#include "dim.cuh"
#include "dim.hxx"
#include "timer.hxx"
#include "cuda_check.cuh"

__constant__ int d_n[3];
__constant__ double d_xmin[3];
__constant__ double d_dx[3];
__constant__ double d_lambda[3];

__global__
void symbol()
{
  printf("symbol : dx = %f %f %f\n", d_dx[0], d_dx[1], d_dx[2]);
  printf("symbol : n  = %d %d %d\n", d_n[0], d_n[1], d_n[2]);
}

void setDims(const int *h_n, 
             const double *h_xmin, 
             const double *h_dx, 
             const double *h_lambda)
{
    Timer & T = GetTimer(5); T.start();
    CUDA_CHECK_OP(hipMemcpyToSymbol(HIP_SYMBOL(d_n), h_n, 3 * sizeof(int)));
    CUDA_CHECK_OP(hipMemcpyToSymbol(HIP_SYMBOL(d_xmin), h_xmin, 3 * sizeof(double)));
    CUDA_CHECK_OP(hipMemcpyToSymbol(HIP_SYMBOL(d_dx), h_dx, 3 * sizeof(double)));
    CUDA_CHECK_OP(hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), h_lambda, 3 * sizeof(double)));
    hipDeviceSynchronize();
    T.stop();
  
//    symbol<<<1,1>>>();  
}
